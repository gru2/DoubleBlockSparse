#include "hip/hip_runtime.h"
#include "ew_op_gpu.h"
#include <stdio.h>
#include "CudaUtils.cuh"
#include <iostream>

// The kernel and the support functions taken form https://github.com/openai/blocksparse

template <typename V>
__global__ void __launch_bounds__(128) gemm_32x32x32_TN_vec4(float* U, const V* __restrict__ X, const V* __restrict__ E, uint C, uint K, uint N, uint C16, uint K16, uint inc_n, uint inc_c, uint inc_k)
{
    __shared__ float shrU[32*32*2 + 16*4];

    uint tid   = threadIdx.x;
    uint idx_C = blockIdx.y;
    uint idx_K = blockIdx.x;
    uint idx_N = blockIdx.z;

    uint tx = tid  & 7;
    uint ty = tid >> 3;
    uint  n = idx_N*32 + ty;

    // global offsets in vector units
    uint c = idx_C*8 + tx;
    uint k = idx_K*8 + tx;
    uint offsetC = n*C + c;
    uint offsetK = n*K + k;

    bool bc = c < C;
    bool bk = k < K;
    //bool bc = true;
    //bool bk = true;

    // shared offsets in bytes
    // When reading, each warp works on its own 8 rows.
    // These groups of 8 are added together at end.
    uint writeS = (ty*32 + tx*4) * 4;
    uint row8   = (tid & 96) * 32;
    uint readCs = row8 + (((tid & 16) >> 3) | (tid & 1)) * 16;
    uint readKs = row8 + ((tid >> 1) & 7) * 16;

    // This keeps all prior logic outside of the loops.
    asm("mov.b32 %0, %0;" : "+r"(writeS)  : );
    asm("mov.b32 %0, %0;" : "+r"(offsetC) : );
    asm("mov.b32 %0, %0;" : "+r"(offsetK) : );
    asm("mov.b32 %0, %0;" : "+r"(readCs)  : );
    asm("mov.b32 %0, %0;" : "+r"(readKs)  : );

    // zero 32 accumulation registers
    float regU[8][4]; // [c][k]
    for (int c = 0; c < 8; c++)
        for (int k = 0; k < 4; k++)
            regU[c][k] = 0;

    // assume a minimum of one loop
    #pragma unroll 1
    do
    {
        V c00, c16;
        V k00, k16;
        ew_zero(c00); ew_zero(c16);
        ew_zero(k00); ew_zero(k16);
        const V* X00 = add_ptr_u(X, offsetC +   0);
        const V* X16 = add_ptr_u(X, offsetC + C16);
        const V* E00 = add_ptr_u(E, offsetK +   0);
        const V* E16 = add_ptr_u(E, offsetK + K16);
        if (bc)
        {
            c00 = __ldg(X00);
            c16 = __ldg(X16);
        }
        if (bk)
        {
            k00 = __ldg(E00);
            k16 = __ldg(E16);
        }
        offsetC += inc_c;
        offsetK += inc_k;
        n       += inc_n;

        __syncthreads();
        st_shared_v4(writeS + ( 0*32 + 0*16*32)*4, to_float(c00));
        st_shared_v4(writeS + ( 0*32 + 1*16*32)*4, to_float(c16));
        st_shared_v4(writeS + (32*32 + 0*16*32)*4, to_float(k00));
        st_shared_v4(writeS + (32*32 + 1*16*32)*4, to_float(k16));
        __syncthreads();

        float regC[8], regK[4];

        #pragma unroll
        for (int j = 0; j < 8; j++)
        {
            // fetch outer product data
            ld_shared_v4(readCs + ( 0*32 + 32*j +  0)*4, &regC[0] );
            ld_shared_v4(readCs + ( 0*32 + 32*j + 16)*4, &regC[4] );
            ld_shared_v4(readKs + (32*32 + 32*j +  0)*4,  regK    );
            // compute outer product
            for (int c = 0; c < 8; c++)
                for (int k = 0; k < 4; k++)
                    regU[c][k] += regC[c] * regK[k];
        }
    } while (n < N);

    // conserve registers by forcing a reload of these
    asm volatile ("mov.u32 %0, %tid.x;"   : "=r"(tid  ) :);
    asm volatile ("mov.u32 %0, %ctaid.x;" : "=r"(idx_K) :);
    asm volatile ("mov.u32 %0, %ctaid.y;" : "=r"(idx_C) :);

    // Arrange 4 tiles horizontally in the X direction: ((tid & 96) >> 2)
    // Add some spacing  to avoid write bank conflicts: (tidY << 2)
    int tidY = ((tid & 16) >> 3) | (tid & 1);
    int tidX = ((tid >> 1) & 7) + ((tid & 96) >> 2) + (tidY << 2);

    float4* storU4 = (float4*)&shrU[tidY*32*4*4 + tidX*4];

    __syncthreads();

    storU4[0*8*4] = *(float4*)regU[0];
    storU4[1*8*4] = *(float4*)regU[1];
    storU4[2*8*4] = *(float4*)regU[2];
    storU4[3*8*4] = *(float4*)regU[3];

    __syncthreads();

    // leaving vector math
    uint tid31 = tid & 31;
    uint tid32 = tid >> 5;
    C *= 4;
    K *= 4;

    float* readU = &shrU[tid32*32*4 + tid31];

    float u[4][4];
    for (int j = 0; j < 4; j++)
        for (int i = 0; i < 4; i++)
            u[j][i] = readU[j*32*4*4 + j*16 + i*32];

    // Tree reduce
    for (int k = 0; k < 4; k++)
        for (int j = 2; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                u[k][i] += u[k][i+j];

    k = idx_K*32 + tid31;
    c = idx_C*32 + tid32;
    bk = k < K;

    uint offsetU = c*K + k;
    atomicRed(add_ptr_u(U, offsetU +  0*K), u[0][0], 0, bk && c +  0 < C);
    atomicRed(add_ptr_u(U, offsetU +  4*K), u[1][0], 0, bk && c +  4 < C);
    atomicRed(add_ptr_u(U, offsetU +  8*K), u[2][0], 0, bk && c +  8 < C);
    atomicRed(add_ptr_u(U, offsetU + 12*K), u[3][0], 0, bk && c + 12 < C);
    //atomicRed(add_ptr_u(U, offsetU +  0*K), u[0][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU +  4*K), u[1][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU +  8*K), u[2][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU + 12*K), u[3][0], 0, true);

    __syncthreads();

    storU4[0*8*4] = *(float4*)regU[4];
    storU4[1*8*4] = *(float4*)regU[5];
    storU4[2*8*4] = *(float4*)regU[6];
    storU4[3*8*4] = *(float4*)regU[7];

    __syncthreads();

    for (int j = 0; j < 4; j++)
        for (int i = 0; i < 4; i++)
            u[j][i] = readU[j*32*4*4 + j*16 + i*32];

    // Tree reduce
    for (int k = 0; k < 4; k++)
        for (int j = 2; j > 0; j >>= 1)
            for (int i = 0; i < j; i++)
                u[k][i] += u[k][i+j];

    atomicRed(add_ptr_u(U, offsetU + 16*K), u[0][0], 0, bk && c + 16 < C);
    atomicRed(add_ptr_u(U, offsetU + 20*K), u[1][0], 0, bk && c + 20 < C);
    atomicRed(add_ptr_u(U, offsetU + 24*K), u[2][0], 0, bk && c + 24 < C);
    atomicRed(add_ptr_u(U, offsetU + 28*K), u[3][0], 0, bk && c + 28 < C);
    //atomicRed(add_ptr_u(U, offsetU + 16*K), u[0][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU + 20*K), u[1][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU + 24*K), u[2][0], 0, true);
    //atomicRed(add_ptr_u(U, offsetU + 28*K), u[3][0], 0, true);
}

bool Gemm_TN(hipStream_t stream, uint SMs,
          float* u,
    const float* xf,
    const float* ef,
    uint C, uint K, uint N)
{
    //hipMemsetD32Async((hipDeviceptr_t)u, 0, C*K, stream);
    hipMemset(u, 0, C*K*sizeof(float)); // TODO use hipMemsetD32Async instead of hipMemset.

    const float4 *x = reinterpret_cast<const float4 *>(xf);
    const float4 *e = reinterpret_cast<const float4 *>(ef);

    uint gridK = CEIL_DIV(K, 32);
    uint gridC = CEIL_DIV(C, 32);
    uint gridN = CEIL_DIV(N, 32);
    C >>= 2;
    K >>= 2;

    // target mult of 6 blocks per SM
    uint smMult = 1, tiles = gridK*gridC;
         if (tiles == 1) smMult = 6;
    else if (tiles <= 4) smMult = 3;
    uint segments = SMs*smMult;
    if (segments > gridN)
        segments = gridN;
    uint seg_len = segments*32;

    dim3 grid(gridK, gridC, segments);
    gemm_32x32x32_TN_vec4<float4><<<grid,128,0,stream>>>(u, x, e, C, K, N, C*16, K*16, seg_len, seg_len*C, seg_len*K);
    return true; // TODO
}
