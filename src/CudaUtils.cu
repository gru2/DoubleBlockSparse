#include <CudaUtils.h>
#include <CudaUtils.cuh>
#include <CudaUtils.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

using namespace CudaUtils;

void cudaErrCheck_(hipError_t stat, const char *file, int line)
{
	if (stat != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
		exit(1);
	}
}

void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line)
{
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
		exit(1);
	}
}

void *deviceAllocate(size_t x)
{
	void *p;
	cudaErrCheck(hipMalloc(&p, x));
	return p;
}

void deviceFree(void *x)
{
	if (x)
	{
		cudaErrCheck(hipFree(x));
	}
}

void memcpyDevice(void *src, void *dst, int len)
{
	cudaErrCheck(hipMemcpy(src, dst, len, hipMemcpyDeviceToDevice));
}

void memcpyDeviceToHost(void *src, void *dst, int len)
{
	cudaErrCheck(hipMemcpy(src, dst, len, hipMemcpyDeviceToHost));
}

void memcpyHostToDevice(void *src, void *dst, int len)
{
	cudaErrCheck(hipMemcpy(src, dst, len, hipMemcpyHostToDevice));
}

CuBlasHandle::CuBlasHandle() : handle(0)
{ }

CuBlasHandle::~CuBlasHandle()
{
	if (!handle)
		return;
	hipblasHandle_t *cuh = static_cast<hipblasHandle_t *>(handle);
	cublasErrCheck(hipblasDestroy(*cuh));
	delete cuh;
}

void *CuBlasHandle::getHandle()
{
	if (handle)
		return;

	hipblasHandle_t *cuh = new cuh;
	cublasErrCheck(hipblasCreate(cuh));
	handle = cuh;
}

void gemm(MatrixF &lhs, MatrixF &rhs, MatrixF &r, CuBlasHandle &handle);
{
	float alpha = 1.0f;
	float beta = 0.0f;
	hipblasHandle_t *cuh = static_cast<hipblasHandle_t *>(handle.getHandle());
	cublasErrCheck(hipblasSgemm(*cuh),
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		rhs.cols, lhs.rows, lhs.cols,
		&alpha,
		rhs.data, rhs.cols,
		lhs.data, lhs.cols,
		&beta,
		r.data, r.cols);
}
