#include "CudaUtils.h"
#include <CudaUtils.cuh>
#include <CudaUtils.h>
#include <stdio.h>
#include <stdlib.h>

void CudaUtils::cudaErrCheck_(hipError_t stat, const char *file, int line)
{
	if (stat != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
		exit(1);
	}
}

void *CudaUtils::deviceAllocate(size_t x)
{
	void *p;
	cudaErrCheck(hipMalloc(&p, x));
	return p;
}

void CudaUtils::deviceFree(void *x)
{
	if (x)
	{
		cudaErrCheck(hipFree(x));
	}
}
