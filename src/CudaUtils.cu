#include <CudaUtils.h>
#include <CudaUtils.cuh>
#include <CudaUtils.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

using namespace CudaUtils;

void CudaUtils::cudaErrCheck_(hipError_t stat, const char *file, int line)
{
	if (stat != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
		exit(1);
	}
}

void CudaUtils::cublasErrCheck_(hipblasStatus_t stat, const char *file, int line)
{
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
		exit(1);
	}
}

void *CudaUtils::deviceAllocate(size_t x)
{
	void *p;
	cudaErrCheck(hipMalloc(&p, x));
	return p;
}

void CudaUtils::deviceFree(void *x)
{
	if (x)
	{
		cudaErrCheck(hipFree(x));
	}
}

void CudaUtils::memcpyDevice(void *dst, void *src, int len)
{
	cudaErrCheck(hipMemcpy(dst, src, len, hipMemcpyDeviceToDevice));
}

void CudaUtils::memcpyDeviceToHost(void *dst, void *src, int len)
{
	cudaErrCheck(hipMemcpy(dst, src, len, hipMemcpyDeviceToHost));
}

void CudaUtils::memcpyHostToDevice(void *dst, void *src, int len)
{
	cudaErrCheck(hipMemcpy(dst, src, len, hipMemcpyHostToDevice));
}

CuBlasHandle::CuBlasHandle() : handle(0)
{ }

CuBlasHandle::~CuBlasHandle()
{
	if (!handle)
		return;
	hipblasHandle_t *cuh = static_cast<hipblasHandle_t *>(handle);
	cublasErrCheck(hipblasDestroy(*cuh));
	delete cuh;
}

void *CuBlasHandle::getHandle()
{
	if (!handle)
	{
		hipblasHandle_t *cuh = new hipblasHandle_t;
		cublasErrCheck(hipblasCreate(cuh));
		handle = cuh;
	}

	return handle;
}

void gemm(MatrixF &lhs, MatrixF &rhs, MatrixF &r, CuBlasHandle &handle)
{
	float alpha = 1.0f;
	float beta = 0.0f;
	hipblasHandle_t *cuh = static_cast<hipblasHandle_t *>(handle.getHandle());
	cublasErrCheck(hipblasSgemm(*cuh,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		rhs.cols, lhs.rows, lhs.cols,
		&alpha,
		rhs.data, rhs.cols,
		lhs.data, lhs.cols,
		&beta,
		r.data, r.cols));
}
