#include "utils.h"

void cudaErrCheck_(hipError_t stat, const char *file, int line)
{
	if (stat != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
		exit(1);
	}
}

void *CudaUtils::deviceAllocate(size_t x)
{
	void *p;
	cudaErrCheck(hipMalloc(&p, x);
	return p;
}

void CudaUtils::deviceFree(void *x)
{
	if (x)
		cudaErrCheck(hipFree(x));
}
